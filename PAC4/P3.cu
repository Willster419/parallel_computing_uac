#include "hip/hip_runtime.h"
#include <stdio.h>

const int blocksize = 4;
const int N=10;
const int periodos=10;

__global__
void moving_average(float *DATA, float *MOV, float *valores)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x, 
  int j=blockIdx.y*blockDim.y + threadIdx.y;
  
  float sum;
  
  for(j=0;j<sizeN-1;j++) {
    sum = 0;
    for(i=0;i<periods;i++) {
      sum += valores[i]
    }
    DATA[i]= sum/periodos;
  }

  for(j==N-1;j++) {
    sum = 0;
    for(i=0;i<N;i++) {
      sum += valores[i]
    }
    MOV[i]= sum/N;
  }
}

int main()
{
  float *DATA_d, *MOV_d;
  int bytes = N*sizeof(float);

  hipMalloc((void**) &DATA_d, bytes);
  hipMalloc((void**) &MOV_d, bytes);
  hipMemcpy(DATA_d, valores, N*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  mavg_device<<<grids, blocks>>> (N, DATA_d, periodos, MOV_d);

  hipMemcpy((void*)out,(void*)MOV_d,bytes,hipMemcpyDeviceToHost);
  hipFree(DATA_d);
  hipFree(MOV_d);

  if(j==N; j++){
    printf("Elemento i = %d j = %d, con Promedio %dn", i, j,MOV[i]);
  }
  else{
    for (j=0; j<N; j++)
      printf("Elementos i = %d j = %d, con Promedio móvil %dn", i, j,DATA[i]);
  }

  return EXIT_SUCCESS;
}